#include "hip/hip_runtime.h"
#include <iostream>
#include <gsm.h>
#include <netcdfcpp.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
using namespace std;

// g++ -I/usr/local/netcdf-c-4.3.2/include -I/usr/local/netcdf-cxx-legacy/include -I/home/jaideep/codes/FIRE_CODES/libgsm_v2/include -L/home/jaideep/codes/FIRE_CODES/libgsm_v2/lib -L/usr/local/netcdf-cxx-legacy/lib -o 1 trend_test.cpp -l:libgsm.so.2 -lnetcdf_c++ 

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void init_residuals_kernel(float * sxx, float * syy, float * sxy, float * sx, float * sy, int count, int nvals){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nvals){
		sxx[i] = sxy[i] = syy[i] = sx[i] = sy[i] = 0;
	}
}

__global__ void update_residuals_kernel(float * vals, float * sxx, float * syy, float * sxy, float * sx, float * sy, int count, int nvals){
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nvals){
	
		sxy[i] += vals[i]*count;
		sx[i] += count;
		sy[i] += vals[i];
		sxx[i] += count*count;
		syy[i] += vals[i] * vals[i];
	
	}
	
}

__global__ void calc_metrics_kernel(float * b1, float * t, float * sxx, float * syy, float * sxy, float * sx, float * sy, int count, int nvals){
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nvals){
		float Sxy = sxy[i] - sx[i]*sy[i]/count; 
		float Sxx = sxx[i] - sx[i]*sx[i]/count;
		b1[i] = Sxy/Sxx;	

		float Syy = syy[i] - sy[i]*sy[i]/count;
		float s = sqrt((Syy - b1[i]*Sxy)/(count-2));

		t[i] = b1[i]/(s/sqrt(Sxx));
	}
	
}


gVar gVar::trend_gpu(double gt1, double gt2){

	clock_t start, end;
	double msecs;
	start = clock();

	int nvals = nlons*nlats*nlevs;
	int blockSize = 128;
	int nblocks = ((nvals-1)/blockSize) + 1;

	float * sxx_dev, * sxy_dev, * syy_dev, *sx_dev, *sy_dev;
	float * b1_dev, *t_dev;
	float * var_dev;

	hipMalloc((void**)& var_dev, nvals*sizeof(float));
	
	hipMalloc((void**)& sxx_dev, nvals*sizeof(float));
	hipMalloc((void**)& sxy_dev, nvals*sizeof(float));
	hipMalloc((void**)& syy_dev, nvals*sizeof(float));
	hipMalloc((void**)& sx_dev, nvals*sizeof(float));
	hipMalloc((void**)& sy_dev, nvals*sizeof(float));
	
	hipMalloc((void**)& b1_dev, nlons*nlats*nlevs*sizeof(float));
	hipMalloc((void**)& t_dev,  nlons*nlats*nlevs*sizeof(float));
	
	gVar temp; temp.copyMeta(*ipvar);
	temp.fill(0);
	int count = 0;
	
	gVar b1 = temp, 
//		 s = temp,
		 t = temp; 
	
	updateInputFile(gt1);	// this will give correct OR one previous file
	while (gt1 > ipvar->ix2gt(ipvar->times.size()-1)){	// increment curr_file as long as gt1 +outside file range
		++curr_file;
		loadInputFileMeta();
	}	
	
	init_residuals_kernel <<< nblocks, blockSize >>> (sxx_dev, syy_dev, sxy_dev, sx_dev, sy_dev, 0, nvals);						

	CDEBUG << "readVar_reduce_mean (" << varname << ") :" << gt2string(gt1) << " " << gt2string(gt2) << endl;
	while(1){
//		cout << (ipvar->times[0]) << " " << (ipvar->times[ipvar->times.size()-1]) << " "  << (gt1-ipvar->tbase)*24.0/ipvar->tscale <<  endl;
		int tstart = lower_bound(ipvar->times.begin(), ipvar->times.end(), (gt1-ipvar->tbase)*24.0/ipvar->tscale) - ipvar->times.begin();	   // first elem >= gt1 
		int tend   = upper_bound(ipvar->times.begin(), ipvar->times.end(), (gt2-ipvar->tbase)*24.0/ipvar->tscale) - ipvar->times.begin() -1;   // last elem <= gt2
//		cout << gt2string(ipvar->ix2gt(tstart)) << " " << gt2string(ipvar->ix2gt(tend)) << endl;

		if (tend < 0) break;

		for (int i=tstart; i<=tend; ++i){ 
			clock_t start1, end1;
			start1 = clock();
			ifile_handle->readVar(*ipvar, i, ipvar->ivar1);	// readCoords() would have set ivar1
			end1 = clock();
			cout << "time to read: " << ((double) (end1 - start1)) * 1000 / CLOCKS_PER_SEC << " ms" << endl;
			
			hipMemcpy(var_dev, &(ipvar->values[0]), ipvar->values.size()*sizeof(float), hipMemcpyHostToDevice);
			gpuErrchk(hipGetLastError());

			update_residuals_kernel <<< nblocks, blockSize >>> (var_dev, sxx_dev, syy_dev, sxy_dev, sx_dev, sy_dev, count, nvals);						
			gpuErrchk(hipGetLastError());
						
			++count;
		}

		if (tend >= ipvar->times.size()-1){ // if tend was the last time in file, then load next file and continue reading
			++curr_file;
			if (curr_file >= filenames.size()) break;
			else loadInputFileMeta();
		}
		else break;
	}

	
	CDEBUG << "----------- Read " << count << " timesteps from " << varname << endl;
	
	calc_metrics_kernel <<< nblocks, blockSize >>> (b1_dev, t_dev, sxx_dev, syy_dev, sxy_dev, sx_dev, sy_dev, count, nvals);						
	
	hipMemcpy(&(b1.values[0]), b1_dev, nvals*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&(t.values[0]),  t_dev,  nvals*sizeof(float), hipMemcpyDeviceToHost);

	end = clock();
	msecs = ((double) (end - start)) * 1000 / CLOCKS_PER_SEC;
	
	cout << "Execution time is " << msecs/1000 << " sec" << endl;
//	if (count > 0) lterpCube(b1, *this, lterp_indices);	// We want to preserve current values if no new values were read
	return b1;
}





int main(){
	
	// ~~~~~~ Some NetCDF Essentials ~~~~~~~~
	// set NETCDF error behavior to non-fatal
	NcError err(NcError::silent_nonfatal);
	
	// specify log file for gsm
	ofstream gsml("gsm_log.txt");
	gsm_log = &gsml;

	// create a grid limits vector for convenience
	float glimits[] = {0, 360, -90, 90};
	vector <float> glim(glimits, glimits+4);
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~

	// create the coordinates for our georeferenced variable
//	int nlons, nlats, nlevs, ntimes;
//	vector <float> lons = createCoord(-180+.5/2,180-0.5/2,0.5,nlons);
//	vector <float> lats = createCoord(-90+.5/2,90-.5/2,0.5,nlats);
//	vector <float> levs = createCoord(1,1,1,nlevs);
//	vector <double> times(16*24); 
//	for (int i=0; i<times.size(); ++i) times[i]= ymd2gday("2001-1-1")+ i*365.2524/24 + 6 - ymd2gday("2000-1-1");

	string files[] = 
	{
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-01-01.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-01-09.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-01-17.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-01-25.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-02-02.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-02-10.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-02-18.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-02-26.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-03-06.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-03-14.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-03-22.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-03-30.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-04-07.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-04-15.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-04-23.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-05-01.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-05-09.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-05-17.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-05-25.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-06-02.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-06-10.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-06-18.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-06-26.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-07-04.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-07-12.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-07-20.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-07-28.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-08-05.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-08-13.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-08-21.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-08-29.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-09-06.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-09-14.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-09-22.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-09-30.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-10-08.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-10-16.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-10-24.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-11-01.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-11-09.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-11-17.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-11-25.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-12-03.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-12-11.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-12-19.nc",
		"/media/jaideep/HD-B1/LAI/LAI_new/MOD15A2H.A_LAI-2003-12-27.nc"
	};

	vector <string> infiles(files, files+46); 
	gVar hires;
	hires.initMetaFromFile(infiles[0]);
	hires.createNcInputStream(infiles, glim);
	hires.printGrid();

	gVar slope = hires.trend_gpu(ymd2gday("2003-1-1"), ymd2gday("2003-12-31"));
	
	slope.writeOneShot("npp.b1.nc");
	
	return 0;

}


