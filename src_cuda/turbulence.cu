#include "hip/hip_runtime.h"
#include <cmath>
#include <fstream>
#include <cstdlib>
#include <string>
#include <hiprand/hiprand_kernel.h>
using namespace std;

#include "../include/turbulence.h"
//#include "../include/graphics.h"
#include "cuda_vector_math.cuh"
#include "cuda_device.h"
#include "../include/histogram.h"


extern hiprandGenerator_t generator_host;


// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// KERNEL to set up RANDOM GENERATOR STATES
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
__global__ void teRngStateSetup_kernel(int * rng_Seeds, hiprandState * rngStates, int nx, int ny){
	//int tid = threadIdx.x;							// each block produces exactly the same random numbers
	int tid_u = threadIdx.x + blockIdx.x*blockDim.x;	// each block produces different random numbers
	if (tid_u >= nx*ny) return;
	
	hiprand_init (rng_Seeds[tid_u], 0, 0, &rngStates[tid_u]);
}

#define TE_PP_SEED 777

void TurbulenceEngine::initRNG(){
	srand(TE_PP_SEED);
	for (int i=0; i<nx*ny; ++i) te_seeds_h[i] = rand(); 
	hipMemcpy( te_seeds_dev, te_seeds_h, sizeof(int)*nx*ny, hipMemcpyHostToDevice);

	int nt = 256, nb = (nx*ny-1)/nt+1;
	teRngStateSetup_kernel <<< nb, nt>>> (te_seeds_dev, te_dev_XWstates, nx, ny);
	getLastCudaError("RNG_kernel_launch");
}

// =========================================================================================
//
//		Turbulence Kernels!!
//
// =========================================================================================


// =========================================================================================
//		Generate conjugate symetric noise matrix
// =========================================================================================
__global__ void te_generateNoise_kernel(float2* Zmat, int nx, int ny, hiprandState * rngStates){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nx*(ny/2+1)) return;
	
	int k = tid % nx;
	int m = tid / nx;	

//	if (m >= ny/2+1) return;
	
	float2 z1 = hiprand_normal2(&rngStates[tid]);
	Zmat[m*nx+k] = z1;

	// conjugate symetric element
	int kc = (nx-k)%nx, mc = (ny-m)%ny;
	Zmat[mc*nx+kc] =  make_float2(z1.x, -z1.y);

}

void TurbulenceEngine::generateNoise_gpu(){
	int nt = 256, nb = (nx*(ny/2+1)-1)/nt+1;
	te_generateNoise_kernel <<< nb, nt>>> (Z_dev, nx, ny, te_dev_XWstates);
	hipMemcpy(Z, Z_dev, nx*ny*sizeof(float2), hipMemcpyDeviceToHost);	

}


// =========================================================================================
//		evolve Psi in fourier domain for 1 time step
// =========================================================================================
__global__ void modifyPsi_kernel(float2 *Z_d, float2 *Psi_d, float *lambda_d, int nx, int ny,
								 float xi, float nu, float dt){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nx*ny) return;
	
	int k = tid % nx;
	int m = tid / nx;	

	int kx = (k > nx/2)? k-nx : k;
	int ky = (m > ny/2)? m-ny : m;

	float normk = kx*kx+ky*ky;
	float sqrt_term;
	if (normk != 0){
		float sqrt_arg = xi*lambda_d[m*nx+k]*(1-exp(-2*nu*normk*dt))/2/nu/normk;
		if (sqrt_arg <0) sqrt_arg = 0;
		sqrt_term = sqrt(sqrt_arg);
	}
	else{ 
		sqrt_term = lambda_d[0]*sqrt(xi*dt);
	}	
	Psi_d[m*nx+k] = Psi_d[m*nx+k]*exp(-nu*normk*dt) + Z_d[m*nx+k]*sqrt_term;

}


void TurbulenceEngine::modifyPsi_gpu(){
	int nt = 256, nb = (nx*ny-1)/nt+1;
	modifyPsi_kernel <<<nb, nt >>> (Z_dev, Psi_dev, lambda_dev, nx, ny, xi, nu, dt);
//	hipMemcpy(Psi, Psi_dev, nx*ny*sizeof(float2), hipMemcpyDeviceToHost);
}


// =========================================================================================
//		calculate velocity field
// =========================================================================================
__global__ void calcVelField_kernel(float2* psi_d, float2* velfield, float L, int nx, int ny){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= (nx-1)*(ny-1)) return;
	
	int ix = tid % (nx-1);
	int iy = tid / (nx-1);	
	
	velfield[iy*(nx-1)+ix].x =  (psi_d[(iy+1)*nx+ ix   ].x - psi_d[iy*nx+ix].x)/(2*L/nx);
	velfield[iy*(nx-1)+ix].y = -(psi_d[    iy*nx+(ix+1)].x - psi_d[iy*nx+ix].x)/(2*L/ny);
	
}

void TurbulenceEngine::calcVelocityField(){
	// calculate velocity field
	int nt = 256, nb = ((nx-1)*(ny-1)-1)/nt+1;
	calcVelField_kernel <<< nb, nt>>> (psi_dev, vel_field_dev, L, nx, ny);

//	hipMemcpy(vel_field, vel_field_dev, (nx-1)*(ny-1)*sizeof(float2), hipMemcpyDeviceToHost);
//	float u[(nx-1)*(ny-1)];
//	for (int i=0; i<(nx-1)*(ny-1); ++i) u[i] = sqrt(vel_field[i].x*vel_field[i].x + vel_field[i].y*vel_field[i].y);
//	float u_max = u[0], u_sum = u[0];
//	for (int i=1; i<(nx-1)*(ny-1); ++i) u_max = fmax(u_max, u[i]);
//	for (int i=1; i<(nx-1)*(ny-1); ++i) u_sum = u_sum + u[i];
//	float u_avg = u_sum/(nx-1)/(ny-1);
//	cout << u_max << " " << u_avg << endl;
}


// =========================================================================================
//
//		Turbulence Engine functions
//
// =========================================================================================


void TurbulenceEngine::init(Initializer &I){
	nx = I.getScalar("nx"); 
	ny = I.getScalar("ny");
	mu = I.getScalar("mu_t"); 
	nu = I.getScalar("nu_t"); 
	xi = I.getScalar("xi_t");
	lambda0 = I.getScalar("lambda0_t");
	dt = I.getScalar("dt_t");
	L = I.getScalar("L");
	xmin = 0; //-I.getScalar("arenaSize")/2;
	xmax = L; // I.getScalar("arenaSize")/2;
	ymin = 0; //-I.getScalar("arenaSize")/2;
	ymax = L; // I.getScalar("arenaSize")/2;

	nlevCol = 64;

	lambda = new float[nx*ny];
	Z = new hipfftComplex[nx*ny];
	Psi = new hipfftComplex[nx*ny];
	psi = new hipfftComplex[nx*ny];
	vel_field = new float2[(nx-1)*(ny-1)];
	te_seeds_h = new int[nx*ny];

	hipMalloc((void**)&lambda_dev, sizeof(float)*nx*ny);
	hipMalloc((void**)&Z_dev, sizeof(hipfftComplex)*nx*ny);
	hipMalloc((void**)&Psi_dev, sizeof(hipfftComplex)*nx*ny);
	hipMalloc((void**)&psi_dev, sizeof(hipfftComplex)*nx*ny);
	hipMalloc((void**)&vel_field_dev, sizeof(float2)*(nx-1)*(ny-1));
	hipMalloc((void**)&te_seeds_dev, nx*ny*sizeof(int));
	hipMalloc((void**)&te_dev_XWstates, nx*ny*sizeof(hiprandState));
	getLastCudaError("alloc GPU arrays");

	// init RNG
	initRNG();

	// prepare to transform
	cout << "creating FFT plan config..." << endl;
	hipfftPlan2d(&plan, nx, ny, HIPFFT_C2C);
	getLastCudaError("create plan");

}


void TurbulenceEngine::freeMemory(){
	delete [] lambda;
	delete [] Z;
	delete [] Psi;
	delete [] psi;
	delete [] vel_field;
	delete [] te_seeds_h;

	hipFree(lambda_dev);
	hipFree(Z_dev);
	hipFree(Psi_dev);
	hipFree(psi_dev);
	hipFree(vel_field_dev);
	hipFree(te_seeds_dev);
	hipFree(te_dev_XWstates);

}

void TurbulenceEngine::generateSpectrum(){
	cout << "Generating spectrum..." << endl;
//	ofstream fout("lambda.txt");
	for (int k=0; k<nx; ++k){
		for (int m=0; m<ny; ++m){
			int x = (k > nx/2)? k-nx : k;
			int y = (m > ny/2)? m-ny : m;
			lambda[m*nx+k] = lambda0*exp(-mu*sqrt(x*x+y*y));
//			fout << lambda[m*nx+k] << " ";
		}
//		fout << "\n";
	}
//	fout.close();
	
	hipMemcpy(lambda_dev, lambda, nx*ny*sizeof(float), hipMemcpyHostToDevice);
}


void TurbulenceEngine::calcEquilPsi(){
	// initial Psi
	for (int k=0; k<nx; ++k){
		for (int m=0; m<ny; ++m){
			int kx = (k > nx/2)? k-nx : k;
			int ky = (m > ny/2)? m-ny : m;

			float normk = kx*kx+ky*ky;
			if (normk != 0) {
				float num = sqrt(xi*lambda[m*nx+k]*(1-exp(-2*nu*normk*dt))/2/nu/normk);
				float den = 1-exp(-nu*normk*dt);
				Psi[m*nx+k] = Z[m*nx+k]*num/den;
			}
			else Psi[m*nx+k] = make_float2(0,0); //Z[m*nx+k];
//			cout << Psi[m*nx+k].x << " ";

			Psi[m*nx+k] = make_float2(1,0);
		}
//		cout << "\n";
	}	

	hipMemcpy(Psi_dev, Psi, nx*ny*sizeof(float2), hipMemcpyHostToDevice);
}


void TurbulenceEngine::transformPsi(){
	hipfftExecC2C(plan, Psi_dev, psi_dev, HIPFFT_BACKWARD);
	getLastCudaError("fft");

	hipDeviceSynchronize();
}


void TurbulenceEngine::normalize_psi(){
	hipMemcpy(psi, psi_dev, nx*ny*sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	
	float tmin=psi[0].x, tmax=psi[0].x, tsum=0;
	for (int i=0; i<nx; ++i){
		for (int j=0; j<ny; ++j){
			tmin = min(tmin, psi[j*nx+i].x);
			tmax = max(tmax, psi[j*nx+i].x);
			tsum += psi[j*nx+i].x;
		}
	}
	float tmean = tsum/nx/ny;
	cout << tmin << " " << tmean << " " << tmax << endl;

	for (int i=0; i<nx; ++i){
		for (int j=0; j<ny; ++j){
			psi[j*nx+i] = (psi[j*nx+i] - tmean)/(tmax-tmin)*2;
		}
	}
	hipMemcpy(psi_dev, psi, nx*ny*sizeof(hipfftComplex), hipMemcpyHostToDevice);
}


void TurbulenceEngine::update(){
	generateNoise_gpu();
	modifyPsi_gpu();
	transformPsi();
	calcVelocityField();
}

void TurbulenceEngine::updateColorMap(){
//	hipMemcpy(psi, psi_dev, nx*ny*sizeof(hipfftComplex), hipMemcpyDeviceToHost);
//	float * tmp = new float[nx*ny];
//	for (int i=0; i<nx*ny; ++i) tmp[i] = psi[i].x;
//	glRenderer->setCmapColorBufferData(tmp, nx*ny, nlevCol);
//	//glutPostRedisplay();
//	delete [] tmp;
}



void TurbulenceEngine::printMap(string mapname, ofstream &fout){
	float * dat = new float[nx*ny];
	for (int iy=0; iy<ny; ++iy){
		for (int ix=0; ix<nx; ++ix){
			if (mapname == "psi") fout << psi[iy*nx+ix].x << " ";
			dat[iy*nx+ix] = psi[iy*nx+ix].x;
		}
		fout << '\n'; 
	}
	fout.flush();

	printSummary(dat, nx*ny);
	delete [] dat;
}


// Particle velocities due to turbulence

//	for (int t=0; t<1; ++t){
//		int ipx = (nx-1)*(ppos.x-xmin)/(xmax-xmin);
//		int ipy = (ny-1)*(ppos.y-ymin)/(ymax-ymin);
//		ppos += vel_field[ipy*(nx-1)+ipx]*dt*0.3;
//		makePeriodic(ppos.x, xmin, xmax);
//		makePeriodic(ppos.y, ymin, ymax);
//		pos_fout << ppos.x << " " << ppos.y << '\n';
//	}



