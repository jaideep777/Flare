#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/gsm.h"
#include <netcdfcpp.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "cuda_device.h"
using namespace std;

// g++ -I/usr/local/netcdf-c-4.3.2/include -I/usr/local/netcdf-cxx-legacy/include -I/home/jaideep/codes/FIRE_CODES/libgsm_v2/include -L/home/jaideep/codes/FIRE_CODES/libgsm_v2/lib -L/usr/local/netcdf-cxx-legacy/lib -o 1 trend_test.cpp -l:libgsm.so.2 -lnetcdf_c++ 

//#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
//inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
//   if (code != hipSuccess){
//      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
//      if (abort) exit(code);
//   }
//}

// This will output the proper error string when calling hipGetLastError
//#define getLastCudaError(ans) { _getLastCudaError(__FILE__, __LINE__, ans); }
//inline void _getLastCudaError(const char *file, int line, string s=""){
//	string errMessage = s;
//	hipError_t err = hipGetLastError();
//	if( err != hipSuccess){
//		cerr << file << "(" << line << ") : Last Cuda Error - " << errMessage 
//			 << " (" << int(err) << "): " << hipGetErrorString(err) << ".\n";
//		exit(-1);
//	}
//}


__global__ void init_residuals_kernel(float * sxx, float * syy, float * sxy, float * sx, float * sy, int count, int nvals){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nvals){
		sxx[i] = sxy[i] = syy[i] = sx[i] = sy[i] = 0;
	}
}

__global__ void update_residuals_kernel(float * vals, float * sxx, float * syy, float * sxy, float * sx, float * sy, int count, int nvals){
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nvals){
	
		sxy[i] += vals[i]*count;
		sx[i] += count;
		sy[i] += vals[i];
		sxx[i] += count*count;
		syy[i] += vals[i] * vals[i];
	
	}
	
}

__global__ void calc_metrics_kernel(float * b1, float * t, float * sxx, float * syy, float * sxy, float * sx, float * sy, int count, int nvals){
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < nvals){
		float Sxy = sxy[i] - sx[i]*sy[i]/count; 
		float Sxx = sxx[i] - sx[i]*sx[i]/count;
		b1[i] = Sxy/Sxx;	

		float Syy = syy[i] - sy[i]*sy[i]/count;
		float s = sqrt((Syy - b1[i]*Sxy)/(count-2));

		t[i] = b1[i]/(s/sqrt(Sxx));
	}
	
}


gVar gVar::trend_gpu(double gt1, double gt2){

	clock_t start, end;
	double msecs;
	start = clock();

	int nvals = nlons*nlats*nlevs;
	int blockSize = 128;
	int nblocks = ((nvals-1)/blockSize) + 1;

	float * sxx_dev, * sxy_dev, * syy_dev, *sx_dev, *sy_dev;
	float * b1_dev, *t_dev;
	float * var_dev;

	hipMalloc((void**)& var_dev, nvals*sizeof(float));
	
	hipMalloc((void**)& sxx_dev, nvals*sizeof(float));
	hipMalloc((void**)& sxy_dev, nvals*sizeof(float));
	hipMalloc((void**)& syy_dev, nvals*sizeof(float));
	hipMalloc((void**)& sx_dev, nvals*sizeof(float));
	hipMalloc((void**)& sy_dev, nvals*sizeof(float));
	
	hipMalloc((void**)& b1_dev, nlons*nlats*nlevs*sizeof(float));
	hipMalloc((void**)& t_dev,  nlons*nlats*nlevs*sizeof(float));
	
	gVar temp; temp.copyMeta(*ipvar);
	temp.fill(0);
	int count = 0;
	
	gVar b1 = temp, 
//		 s = temp,
		 t = temp; 
	
	updateInputFile(gt1);	// this will give correct OR one previous file
	while (gt1 > ipvar->ix2gt(ipvar->times.size()-1)){	// increment curr_file as long as gt1 +outside file range
		++curr_file;
		loadInputFileMeta();
	}	
	
	init_residuals_kernel <<< nblocks, blockSize >>> (sxx_dev, syy_dev, sxy_dev, sx_dev, sy_dev, 0, nvals);						

	CDEBUG << "readVar_reduce_mean (" << varname << ") :" << gt2string(gt1) << " " << gt2string(gt2) << endl;
	while(1){
		int tstart = lower_bound(ipvar->times.begin(), ipvar->times.end(), (gt1-ipvar->tbase)*24.0/ipvar->tscale) - ipvar->times.begin();	   // first elem >= gt1 
		int tend   = upper_bound(ipvar->times.begin(), ipvar->times.end(), (gt2-ipvar->tbase)*24.0/ipvar->tscale) - ipvar->times.begin() -1;   // last elem <= gt2

		if (tend < 0) break;

		for (int i=tstart; i<=tend; ++i){ 
			ifile_handle->readVar(*ipvar, i, ipvar->ivar1);	// this is the slowest step which will run in parallel with the kernel execution. Hence no need to parallelize GPU-IO and kernel execution 
			
			hipMemcpy(var_dev, &(ipvar->values[0]), ipvar->values.size()*sizeof(float), hipMemcpyHostToDevice);
			getLastCudaError("memcpy"); //hipGetLastError());

			update_residuals_kernel <<< nblocks, blockSize >>> (var_dev, sxx_dev, syy_dev, sxy_dev, sx_dev, sy_dev, count, nvals);						
			getLastCudaError("kernel"); //hipGetLastError());
						
			++count;
		}

		if (tend >= ipvar->times.size()-1){ // if tend was the last time in file, then load next file and continue reading
			++curr_file;
			if (curr_file >= filenames.size()) break;
			else loadInputFileMeta();
		}
		else break;
	}

	
	CDEBUG << "----------- Read " << count << " timesteps from " << varname << endl;
	
	calc_metrics_kernel <<< nblocks, blockSize >>> (b1_dev, t_dev, sxx_dev, syy_dev, sxy_dev, sx_dev, sy_dev, count, nvals);						
	
	hipMemcpy(&(b1.values[0]), b1_dev, nvals*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&(t.values[0]),  t_dev,  nvals*sizeof(float), hipMemcpyDeviceToHost);

	end = clock();
	msecs = ((double) (end - start)) * 1000 / CLOCKS_PER_SEC;

	hipFree(var_dev);
	hipFree(sxx_dev);
	hipFree(sxy_dev);
	hipFree(syy_dev);
	hipFree(sx_dev);
	hipFree(sy_dev);
	hipFree(b1_dev);
	hipFree(t_dev);
	
	cout << "Execution time is " << msecs/1000 << " sec" << endl;
//	if (count > 0) lterpCube(b1, *this, lterp_indices);	// We want to preserve current values if no new values were read
	return b1;
}



/*
int main(){
	
	// ~~~~~~ Some NetCDF Essentials ~~~~~~~~
	// set NETCDF error behavior to non-fatal
	NcError err(NcError::silent_nonfatal);
	
	// specify log file for gsm
	ofstream gsml("gsm_log.txt");
	gsm_log = &gsml;

	// create a grid limits vector for convenience
	float glimits[] = {0, 360, -90, 90};
	vector <float> glim(glimits, glimits+4);
	// ~~~~~~~~~~~~~~~~~~~~~~~~~~

	// create the coordinates for our georeferenced variable
//	int nlons, nlats, nlevs, ntimes;
//	vector <float> lons = createCoord(-180+.5/2,180-0.5/2,0.5,nlons);
//	vector <float> lats = createCoord(-90+.5/2,90-.5/2,0.5,nlats);
//	vector <float> levs = createCoord(1,1,1,nlevs);
//	vector <double> times(16*24); 
//	for (int i=0; i<times.size(); ++i) times[i]= ymd2gday("2001-1-1")+ i*365.2524/24 + 6 - ymd2gday("2000-1-1");

	string files[] = 
	{
		"/media/jaideep/WorkData/Fire_G/GPP_modis/gpp.2000-2015.nc",
	};

	vector <string> infiles(files, files+1); 
	gVar hires;
	hires.initMetaFromFile(infiles[0]);
	hires.createNcInputStream(infiles, glim);
	hires.printGrid();

	gVar slope = hires.trend_gpu(ymd2gday("2000-1-1"), ymd2gday("2015-12-31"));
	
	slope.writeOneShot("npp.t1.nc");
	
	return 0;

}

*/



