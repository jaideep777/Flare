#include "hip/hip_runtime.h"
#include <cmath>
#include <fstream>
#include <cstdlib>
#include <string>
#include <hiprand/hiprand_kernel.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

using namespace std;

#include "../include/resource.h"
//#include "../headers/graphics.h"
#include "cuda_vector_math.cuh"
#include "cuda_device.h"



// extern hiprandGenerator_t generator_host;

// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// KERNEL to set up RANDOM GENERATOR STATES
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
//__global__ void teRngStateSetup_kernel(int * rng_Seeds, hiprandState * rngStates, int nx, int ny){
//	//int tid = threadIdx.x;							// each block produces exactly the same random numbers
//	int tid_u = threadIdx.x + blockIdx.x*blockDim.x;	// each block produces different random numbers
//	if (tid_u >= nx*ny) return;
//	
//	hiprand_init (rng_Seeds[tid_u], 0, 0, &rngStates[tid_u]);
//}

//#define TE_PP_SEED time(NULL)

//void ResourceGrid::initRNG(){
//	srand(TE_PP_SEED);
//	for (int i=0; i<nx*ny; ++i) te_seeds_h[i] = rand(); 
//	hipMemcpy( te_seeds_dev, te_seeds_h, sizeof(int)*nx*ny, hipMemcpyHostToDevice);

//	int nt = 256, nb = (nx*ny-1)/nt+1;
//	teRngStateSetup_kernel <<< nb, nt>>> (te_seeds_dev, te_dev_XWstates, nx, ny);
//	getLastCudaError("RNG_kernel_launch");
//}



// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
//          RESOURCE GRID
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~


void ResourceGrid::init(Initializer &I){
	nx = I.getScalar("nx"); 
	ny = I.getScalar("ny");
	D  = I.getScalar("D");
	dt = I.getScalar("dt");
	L  = I.getScalar("L");

	graphics = I.getScalar("graphicsQual")>0;

	res = new float[nx*ny];
	hipMalloc((void**)&res_dev, sizeof(float)*nx*ny);
	hipMalloc((void**)&res_new_dev, sizeof(float)*nx*ny);


	r = new float[nx*ny];
	K = new float[nx*ny];

	float r0 = I.getScalar("r");
	float K0 = I.getScalar("K");
	for (int i=0; i<nx*ny; ++i){
		r[i] = r0;
		K[i] = K0;
	}
	hipMalloc((void**)&r_dev, sizeof(float)*nx*ny);
	hipMalloc((void**)&K_dev, sizeof(float)*nx*ny);

	hipMemcpy(r_dev, r, nx*ny*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(K_dev, K, nx*ny*sizeof(float), hipMemcpyHostToDevice);	
	

	for (int i=0; i<nx*ny; ++i) res[i]=K0;
	res[ix2(128,128,256)] = K0;
	
	hipMemcpy(res_dev, res, nx*ny*sizeof(float), hipMemcpyHostToDevice);

//	if (graphics){
//		// create resource grid color-map
//		res_shape = ColorMap("res", false, 100, nx, 0, L);
//		float2 cmap_pos[res_shape.nVertices];
//		res_shape.createGridCentres(cmap_pos); 
//		res_shape.createShaders();
//		res_shape.createVBO(cmap_pos, res_shape.nVertices*sizeof(float2));	
//		res_shape.createColorBuffer();
//		res_shape.updateColors(res, nx*ny);
//	}

	cout << "total resource = " << sumResource() << endl;

}


void ResourceGrid::freeMemory(){
	delete [] res;
	delete [] r;
	delete [] K;
	hipFree(res_dev);
	hipFree(res_new_dev);
	hipFree(r_dev);
	hipFree(K_dev);
	
//	if (graphics){
//		res_shape.deleteShaders();
//		res_shape.deleteVBO();
//	}
}


//void ResourceGrid::graphics_updateArrays(){
//	hipMemcpy(res, res_dev, nx*ny*sizeof(float), hipMemcpyDeviceToHost);
//	res_shape.updateColors(res, nx*ny, 0, 50);
//}



// =========================================================================================
//
//		Resource dynamics Kernels
//
// =========================================================================================


__global__ void diffusion_kernel(float * res, float * res_new, float D, int nx, int ny, float dt){

	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nx*ny) return;

	int ix = tid % nx;
	int iy = tid / nx;
	
	float grad_x = res[ix2(makePeriodicID(ix+1,nx),                iy,       nx)]
				 + res[ix2(makePeriodicID(ix-1,nx),                iy,       nx)]
				 - res[ix2(               ix,                      iy,       nx)]*2;
	float grad_y = res[ix2(               ix,       makePeriodicID(iy+1,ny), nx)]
				 + res[ix2(               ix,       makePeriodicID(iy-1,ny), nx)]
				 - res[ix2(               ix,                      iy,       nx)]*2;

	res_new[tid] = res[tid] + (D*grad_x+D*grad_y)*dt;	

}


void ResourceGrid::diffuse(){
	int nt = 256; int nb = (nx*ny-1)/nt + 1;

	diffusion_kernel <<<nb, nt>>> (res_dev, res_new_dev, D, nx, ny, dt);
	hipMemcpy(res_dev, res_new_dev, nx*ny*sizeof(float), hipMemcpyDeviceToDevice);
}


__global__ void resource_growth_kernel(float * res, float *r, float *Ke_all, float *K, float dt, int nx, int ny){
	
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nx*ny) return;

	float R = res[tid];
	R += ( r[tid]*R*(1-R/K[tid]) - Ke_all[tid]*R )*dt;  // resource is extracted after growth
	res[tid] = fmaxf(1e-6, R);	// resource should not go negative
}

void ResourceGrid::grow(float * ke_all_dev){
	int nt = 256; int nb = (nx*ny-1)/nt + 1;
	resource_growth_kernel <<<nb, nt>>> (res_dev, r_dev, ke_all_dev, K_dev, dt, nx, ny);
	getLastCudaError("resource growth kernel");
}


float ResourceGrid::sumResource(){
	thrust::device_ptr <float> arr_dev(res_dev);
	totalRes = thrust::reduce(arr_dev, arr_dev+nx*ny);
	return totalRes;
}



//void ResourceGrid::update(){
//	grow();
//	//diffuse();
//}



